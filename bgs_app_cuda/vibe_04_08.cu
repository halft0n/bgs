#include "hip/hip_runtime.h"
#include "vibe.h"
#include <hip/hip_runtime.h> 
#include <opencv2/core/core.hpp>
#include <random>
#include <time.h>
#include <iostream>
#include <fstream>

#define thread_num_per_block 256
#define block_num 128

extern int cnt;
std::ofstream fout("D:\\cuda.txt");

inline void checkCudaErrors(hipError_t err)//��������
{
	if (hipSuccess != err)
	{
	    fprintf(stderr, "CUDA Runtime API error: %s.\n", hipGetErrorString(err));
	    return;
	}
}

__device__ int dev_max(int i, int j)
{
	if(i>j)
		return i;
	else
		return j;
}

__device__ int dev_min(int i, int j)
{
	if(i<j)
		return i;
	else
		return j;
}

__device__ void dev_getRndNeighbor(int i, int j, unsigned int *dev_rng, int width,
								int height, int* rng_idx, int pixel_neighbor_,
								int *rnd_pos)
{
	int neighbor_count = (pixel_neighbor_ * 2 + 1) * (pixel_neighbor_ * 2 + 1);
	*rng_idx = (*rng_idx + 1) % RANDOM_BUFFER_SIZE;
	int rnd = dev_rng[*rng_idx] % neighbor_count;
	int start_i = i - pixel_neighbor_;
	int start_j = j - pixel_neighbor_;
	int area = pixel_neighbor_ * 2 + 1;
	int position_i = rnd / area;
	int position_j = rnd % area;

	int cur_i = dev_max(dev_min(start_i + position_i, height - 1), 0);
	int cur_j = dev_max(dev_min(start_j + position_j, width - 1), 0);
	rnd_pos[0] = cur_i;
	rnd_pos[1] = cur_j;

}

__global__ void init_cuda(unsigned char* model, unsigned char* image, 
					unsigned int* rng, int width, int height, 
					int channels, int samples, 
					int pixel_neighbor, int* rng_idx)
{
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	long image_length = width * height;

	for (size_t k=tid;k<image_length;k+=block_num*thread_num_per_block)
	{
		int i = k/width;
		int j = k%width;

		int rnd_pos[2];
		for (int c = 0; c < channels; c++) 
		{
			model[channels * samples * width * i +
					channels * samples * j + c] =
				image[channels * width * i + channels * j + c];
		}
		for (int s = 1; s < samples; s++) 
		{
			dev_getRndNeighbor(i, j, rng, width, height, rng_idx, pixel_neighbor, rnd_pos);
			int img_idx =
			    channels * width * rnd_pos[0] + channels * rnd_pos[1];
			int model_idx = channels * samples * width * i +
                        channels * samples * j + channels * s;
			for (int c = 0; c < channels; c++) 
			{
			    model[model_idx + c] = image[img_idx + c];
		    }
		}
	}
}

__global__ void update_cuda(unsigned char* model, unsigned char* image,
					unsigned char* mask, unsigned int* rng,
					int width, int height,int pixel_neighbor,
					int distance_threshold, int matching_threshold,
					int update_factor, int channels, int samples,
					int* rng_idx )
{
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	long image_length = width * height;

	int added_num = block_num*thread_num_per_block;
	for (size_t k=tid;k<image_length;k+=added_num)
	{
		int i = k/width;
		int j = k%width;

		bool flag = false;
		int matching_counter = 0;
		int img_idx = width * i + j;

		for(int s=0; s<samples; s++)
		{
			int model_idx = samples * (width * i + j) + s;
			if(std::abs(image[img_idx] - model[model_idx])
								< distance_threshold)
				matching_counter++;
			//����ģ������һ֡ͼ��ƥ�����������ֵ�����ص㣨i��j��Ϊ������
			if (matching_counter > matching_threshold) 
			{
				flag = true;
				break;
			}
		}
		
		if (flag) 
		{
			mask[width * i + j] = 0;//��ɫ
			*rng_idx = ((*rng_idx) + 1) % RANDOM_BUFFER_SIZE;
			//����һ�����ʱ���ģ�ͣ���������update_factor
			if (rng[(*rng_idx)] % update_factor) 
			{
				*rng_idx = ((*rng_idx) + 1) % RANDOM_BUFFER_SIZE;
				int sample = rng[(*rng_idx)] % samples;
				int model_idx = samples *width * i +
						  samples * j + sample;
				//���µ�ǰ���ص�(x,y)�ı���ģ��
				model[model_idx] = image[img_idx];
				//�����������ѡȡһ��������ı���ģ��
				int rnd_pos[2];
				dev_getRndNeighbor(i, j, rng, width, height, rng_idx,
					pixel_neighbor, rnd_pos);
				*rng_idx = ((*rng_idx) + 1) % RANDOM_BUFFER_SIZE;
				sample = rng[(*rng_idx)] % samples;
				model_idx = samples * width * rnd_pos[0] +
					samples * rnd_pos[1] + sample;
				model[model_idx] = image[img_idx];
			}
		} else 
		{
			mask[width * i + j] = 255;//��ɫ
		}
	}
}

namespace masa_video_compression {

VIBE::VIBE(int channels, int samples, int pixel_neighbor,
           int distance_threshold, int matching_threshold, int update_factor)
    : samples_(samples), channels_(channels), pixel_neighbor_(pixel_neighbor),
      distance_threshold_(distance_threshold),
      matching_threshold_(matching_threshold), update_factor_(update_factor) {

  model_ = nullptr;
  rng_idx_ = 0;
  srand(0);
  for (int i = 0; i < RANDOM_BUFFER_SIZE; i++) {
    rng_[i] = rand();
  }
}

VIBE::~VIBE() {
  if (model_ != nullptr) {
    delete[] model_;
  }
    hipFree(dev_image);//�����Կ��ڴ�
	hipFree(dev_model);
	hipFree(dev_mask);
	hipFree(dev_rng);
	hipFree(dev_rng_idx);
}

void VIBE::init(const cv::Mat &img) {
  CV_Assert(img.channels() == channels_);
  size_ = img.size();
  model_ = new unsigned char[channels_ * size_.width * size_.height * samples_];
  mask_ = cv::Mat(size_, CV_8UC1, cv::Scalar::all(0));

	width = img.size().width;
	height = img.size().height;
	fout<<"video information:"<<width<<"*"<<height<<std::endl;

	image_size = width * height;
	model_size = channels_ * image_size * samples_;

	long init_start  = clock();
	//�����Կ��ڴ�
	checkCudaErrors(hipMalloc((void**)&dev_image, sizeof(unsigned char)* 
		channels_ * image_size ));
	checkCudaErrors(hipMalloc((void**)&dev_model, sizeof(unsigned char)* 
		model_size));
	checkCudaErrors(hipMalloc((void**)&dev_mask, sizeof(unsigned char)* 
		channels_ * image_size ));
	checkCudaErrors(hipMalloc((void**)&dev_rng, sizeof(unsigned int)* 
		RANDOM_BUFFER_SIZE));
	checkCudaErrors(hipMalloc((void**)&dev_rng_idx, sizeof(unsigned int)));
	
	//�����������������ڴ�鸴�Ƶ��Կ��ڴ���
	checkCudaErrors(hipMemcpy(dev_image, img.data, sizeof(unsigned char)* 
				channels_ * image_size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_mask, mask_.data, sizeof(unsigned char)* 
				image_size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_rng, rng_, sizeof(unsigned int)* 
				RANDOM_BUFFER_SIZE, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_rng_idx, &rng_idx_, sizeof(unsigned int),
				hipMemcpyHostToDevice));

	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	init_cuda<<<block_num, thread_num_per_block>>>//�����Կ���������
			(dev_model, dev_image, dev_rng, width, height, channels_, 
			samples_, pixel_neighbor_, dev_rng_idx);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);

	fout<<"init time:"<<msecTotal1<<std::endl;
}

void VIBE::update(const cv::Mat &img) {
  CV_Assert(channels_ == img.channels() && size_ == img.size());
  
	

	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);
	checkCudaErrors(hipMemcpy(dev_image, img.data, sizeof(unsigned char)* 
				channels_ * image_size, hipMemcpyHostToDevice));
	update_cuda<<<block_num, thread_num_per_block>>>//�����Կ���������
		(	dev_model, dev_image, dev_mask, dev_rng, width, height, pixel_neighbor_, 
			distance_threshold_, matching_threshold_, update_factor_, channels_, 
			samples_, dev_rng_idx);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);

	fout<<"NO."<<cnt<<" time: "<<msecTotal1<<"ms\n";
}

cv::Mat &VIBE::getMask()
{ 
	checkCudaErrors(hipMemcpy(mask_.data, dev_mask, sizeof(unsigned char)*
		image_size, hipMemcpyDeviceToHost));
	return mask_;
}

}
